#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#define BLOCK_SIZE 16

__global__ void vectorAdd(int *a, int *b, int *c, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n)
        c[idx] = a[idx] + b[idx];
}

__global__ void matrixMul(int *a, int *b, int *c, int m, int k, int n){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(row < m && col < n){
        int sum = 0;
        for(int i = 0; i < k; i++){
            sum += a[row * k + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

void fillIntVector(int *arr, int size){
    for(int i = 0; i < size; i++)arr[i] = rand() % 100;
}

int main(){
    srand(time(0));

    // Vector Addition

    int size;
    cout<<"Enter vector size: ";
    cin>>size;

    int *h_vecA = new int[size];
    int *h_vecB = new int[size];
    int *h_vecC = new int[size];

    fillIntVector(h_vecA, size);
    fillIntVector(h_vecB, size);

    int *d_vecA, *d_vecB, *d_vecC;
    hipMalloc(&d_vecA, size * sizeof(int));
    hipMalloc(&d_vecB, size * sizeof(int));
    hipMalloc(&d_vecC, size * sizeof(int));

    hipMemcpy(d_vecA, h_vecA, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vecB, h_vecB, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((size + block.x - 1) / block.x);

    hipEvent_t startVec, stopVec;
    hipEventCreate(&startVec);
    hipEventCreate(&stopVec);
    hipEventRecord(startVec);

    vectorAdd<<<grid, block>>>(d_vecA, d_vecB, d_vecC, size);
    hipDeviceSynchronize();

    hipEventRecord(stopVec);
    hipEventSynchronize(stopVec);
    float vectorAddTime;
    hipEventElapsedTime(&vectorAddTime, startVec, stopVec);

    hipMemcpy(h_vecC, d_vecC, size * sizeof(int), hipMemcpyDeviceToHost);

    cout<<"Time taken for vector addition: "<<vectorAddTime<<"ms\n";
    cout<<"Resultant Vector: \n";
    for(int i = 0; i < size; i++)cout<<h_vecC[i]<<" ";
    cout<<endl;

// Matrix Multiplication

    cout<<"Matrix 1 Dimension (m x k) \nMatrix 2 Dimension (k x n) \nEnter value of m, k, n: ";
    int m,k,n;
    cin>>m>>k>>n;

    int *h_matA = new int[m*k]; 
    int *h_matB = new int[k*n];
    int *h_matC = new int[m*n];

    fillIntVector(h_matA, m*k);
    fillIntVector(h_matB, k*n);

    int *d_matA, *d_matB, *d_matC;
    hipMalloc(&d_matA, m*k*sizeof(int));
    hipMalloc(&d_matB, k*n*sizeof(int));
    hipMalloc(&d_matC, m*n*sizeof(int));

    hipMemcpy(d_matA, h_matA, m*k*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_matB, h_matB, k*n*sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((n + BLOCK_SIZE - 1)/ BLOCK_SIZE, (m + BLOCK_SIZE - 1)/ BLOCK_SIZE);

    hipEvent_t matstart, matstop;
    hipEventCreate(&matstart);
    hipEventCreate(&matstop);
    hipEventRecord(matstart);

    matrixMul<<<gridDim, blockDim>>>(d_matA, d_matB, d_matC, m, k, n);
    hipDeviceSynchronize();

    hipEventRecord(matstop);
    hipEventSynchronize(matstop);
    float mat_time;
    hipEventElapsedTime(&mat_time, matstart, matstop);

    hipMemcpy(h_matC, d_matC, m*n*sizeof(int), hipMemcpyDeviceToHost);

    cout<<"Time taken for matrix multiplication: "<<mat_time<<"ms\n";
    cout<<"Resultant Matrix:\n";
    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++)cout<<h_matC[i * n + j]<<" ";
        cout<<endl;
    }

    hipFree(d_vecA);
    hipFree(d_vecB);
    hipFree(d_vecC);
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);
    delete[] h_vecA;
    delete[] h_vecB;
    delete[] h_vecC;
    delete[] h_matA;
    delete[] h_matB;
    delete[] h_matC;
}
// nvcc cuda.cu -o cuda -lcudart -lcublas -lcublasLt -lcudnn -lcurand
// nvcc cuda.cu -o cuda
// ./cuda